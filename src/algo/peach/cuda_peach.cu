#include "hip/hip_runtime.h"
/* cuda_peach.cu   Multi-GPU CUDA Mining
 *
 * Copyright (c) 2019 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 *
 * Date: 10 July 2019
 * Revised: 22 July 2019
 * Revision: 3
 *
 * Optimized version, cloning from cuda_peach.cu.
 *
 */

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <nvml.h>

#include "../../crypto/hash/cpu/sha256.c"
#include "../../config.h"
#include "peach.h"
#include "nighthash.cu"
#include "cuda_peach.h"

__constant__ static CUDA_SHA256_CTX __align__(8) c_precomputed_sha256;
__constant__ static uint8_t __align__(8) c_phash[32];
__constant__ static uint8_t __align__(8) c_input[16384];
__constant__ static uint8_t __align__(8) c_difficulty;


inline int cudaCheckError( const char *msg, uint32_t gpu, const char *file)
{
   hipError_t err = hipGetLastError();
   if(hipSuccess != err) {
      fprintf(stderr, "%s Error (#%d) in %s: %s\n",
              msg, gpu, file, hipGetErrorString(err));
      return 1;
   }
   return 0;
}


__device__ uint32_t cuda_next_index(uint32_t index, uint8_t *g_map,
                                    uint8_t *first_seed, uint8_t *last_seed)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[HASHLEN + 4 + TILE_LENGTH];
   byte hash[HASHLEN];
   int i, seedlen;

   /* Create nighthash seed for this index on the map */
   seedlen = HASHLEN + 4 + TILE_LENGTH;
   
   memcpy(seed, first_seed, 16);
   memcpy(seed + 16, last_seed, 16);
   memcpy(seed + HASHLEN, &index, 4);
   memcpy(seed + 36, &g_map[index * TILE_LENGTH], TILE_LENGTH);

   /* Setup nighthash the seed, NO TRANSFORM */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 0);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, hash);

   /* Convert 32-byte Hash Value Into 8x 32-bit Unsigned Integer */
   for(i = 0, index = 0; i < 8; i++)
      index += ((uint32_t *) hash)[i];

   return index & (MAP-1);
}


__device__ void cuda_gen_tile(uint32_t index, uint8_t *g_map)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[4 + HASHLEN];
   byte *tilep;
   int i, j, seedlen;

   /* Set map pointer */
   tilep = &g_map[index * TILE_LENGTH];

   /* Create nighthash seed for this index on the map */
   seedlen = 4 + HASHLEN;
   
   memcpy(seed, &index, 4);
   memcpy(seed + 4, c_phash, 32);

   /* Setup nighthash with a transform of the seed */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 1);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, tilep);

   /* Begin constructing the full tile */
   for(i = 0; i < TILE_LENGTH; i += HASHLEN) { /* For each tile row */
      /* Set next row's pointer location */
      j = i + HASHLEN;

      /* Hash the current row to the next, if not at the end */
      if(j < TILE_LENGTH) {
         /* Setup nighthash with a transform of the current row */
         cuda_nighthash_init(&nighthash, &tilep[i], HASHLEN, index, 1);

         /* Update nighthash with the seed data and tile index */
         cuda_nighthash_update(&nighthash, &tilep[i], HASHLEN);
         cuda_nighthash_update(&nighthash, (byte *) &index, 4);

         /* Finalize nighthash into the first 32 byte chunk of the tile */
         cuda_nighthash_final(&nighthash, &tilep[j]);
      }
   }
}


__global__ void cuda_build_map(uint8_t *g_map)
{
   const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
   if (thread < MAP)
      cuda_gen_tile(thread, g_map);
}


__global__ void cuda_find_peach(uint32_t offset, uint8_t *g_map,
                                int32_t *g_found, uint8_t *g_nonce)
{
   __shared__
      uint8_t first_seed[16];
      uint8_t last_seed[16], hash[32];
   CUDA_SHA256_CTX ictx;
   int32_t i, j, n;
   uint32_t sm;
   
   /******************/
   /* Assemble seeds */
   if(threadIdx.x < 16)
      first_seed[threadIdx.x] = c_input[((offset + blockIdx.x) << 4) + threadIdx.x];
   
   __syncthreads();
   
   memcpy(last_seed, &c_input[threadIdx.x << 4], 16);
   
   /*********************************************************/
   /* Hash 124 bytes of Block Trailer, including both seeds */
   memcpy(&ictx, &c_precomputed_sha256, sizeof(CUDA_SHA256_CTX));
   
   cuda_sha256_update(&ictx, first_seed, 16);
   cuda_sha256_update(&ictx, last_seed, 16);
   cuda_sha256_final(&ictx, hash);
   
   /****************************************************/
   /* Follow the tile path based on the selected nonce */
   sm = hash[0];
   #pragma unroll
   for(i = 1; i < HASHLEN; i++)
       sm *= hash[i];
   sm = (sm & (MAP - 1));
   
   /* make <JUMP> tile jumps to find the final tile */
   #pragma unroll
   for(j = 0; j < JUMP; j++)
      sm = cuda_next_index(sm, g_map, first_seed, last_seed);
   
   /****************************************************************/
   /* Check the hash of the final tile produces the desired result */
   cuda_sha256_init(&ictx);
   cuda_sha256_update(&ictx, hash, HASHLEN);
   cuda_sha256_update(&ictx, &g_map[sm * TILE_LENGTH], TILE_LENGTH);
   cuda_sha256_final(&ictx, hash);
   
   /* Evaluate hash */
   for(i = j = n = 0; i < 8 && n == j; i++, j += 32)
      n = __clz( __byte_perm( ((int32_t *) hash)[i], 0, 0x0123 ) );

   if(n >= c_difficulty && !atomicExch(g_found, 1)) {
      /* PRINCESS FOUND! */
      memcpy(g_nonce, first_seed, 16);
      memcpy(g_nonce + 16, last_seed, 16);
   }
   /* Our princess is in another castle ! */
}


extern "C" {


uint8_t enable_nvml = 0;
GPU_t gpus[MAX_GPUS] = { 0 };
uint32_t num_gpus = 0;
/* Max 63 GPUs Supported */
PeachCudaCTX peach_ctx[64];
PeachCudaCTX *ctx = peach_ctx;
int32_t nGPU = 0;
SHA256_CTX *precompute_ctx;
int32_t *found;
byte *diff;
byte *phash;

int init_nvml() {
   int32_t num_cuda = 0;
   hipError_t cr = hipGetDeviceCount(&num_cuda);
   if (num_cuda > MAX_GPUS) num_cuda = MAX_GPUS;

   for (int i = 0; i < num_cuda; i++) {
      struct hipDeviceProp_t p = { 0 };
      hipError_t cr = hipGetDeviceProperties(&p, i);
      printf("CUDA pciDomainID: %x, pciBusID: %x, pciDeviceID: %x\n", p.pciDomainID, p.pciBusID, p.pciDeviceID);
      gpus[i].pciDomainId = p.pciDomainID;
      gpus[i].pciBusId = p.pciBusID;
      gpus[i].pciDeviceId = p.pciDeviceID;
      gpus[i].cudaNum = i;
      num_gpus++;
   }


   nvmlReturn_t r = nvmlInit();
   if (r != NVML_SUCCESS) {
      printf("Failed to initialize NVML: %s\n", nvmlErrorString(r));
      enable_nvml = 0;
      return 0;
   }
   uint32_t nvml_device_count;
   r = nvmlDeviceGetCount(&nvml_device_count);
   if (r != NVML_SUCCESS) {
      printf("Failed to get NVML device count: %s\n", nvmlErrorString(r));
      enable_nvml = 0;
      return 0;
   }
   printf("NVML Devices: %d\n", nvml_device_count);
   for (int i = 0; i < nvml_device_count; i++) {
      nvmlDevice_t dev;
      r = nvmlDeviceGetHandleByIndex(i, &dev);
      if (r != NVML_SUCCESS) {
         printf("nvmlDeviceGetHandleByIndex failed: %s\n", nvmlErrorString(r));
         nvml_device_count = i;
         break;
      }
      nvmlPciInfo_t pci;
      r = nvmlDeviceGetPciInfo(dev, &pci);
      if (r != NVML_SUCCESS) {
         printf("nvmlDeviceGetPciInfo failed: %s\n", nvmlErrorString(r));
         continue;
      }
      printf("NVML PCI: pciDeviceId: %x, pciSubSystemId: %x, domain: %x, device: %x, bus: %x\n", pci.pciDeviceId, pci.pciSubSystemId, pci.domain, pci.device, pci.bus);

      for (int j = 0; j < num_cuda; j++) {
         if (gpus[j].pciDomainId == pci.domain && gpus[j].pciBusId == pci.bus && gpus[i].pciDeviceId == pci.device) {
            printf("NVML device is CUDA Device: %d\n", gpus[j].cudaNum);
            gpus[j].nvml_dev = dev;
            break;
         }
      }

      char device_name[128];
      r = nvmlDeviceGetName(dev, device_name, 128);
      if (r != NVML_SUCCESS) {
         printf("nvmlDeviceGetName failed: %s\n", nvmlErrorString(r));
      }
      else {
         printf("Device: %d, Name: %s\n", i, device_name);
      }
   }
   enable_nvml = 1;
   return 1;
}

int init_cuda_peach(byte difficulty, byte *prevhash, byte *bt) {
   int i;
   
   /* Obtain and check system GPU count */
   nGPU = 0;
   hipGetDeviceCount(&nGPU);
   if(nGPU<1 || nGPU>64) return nGPU;
   
   /* Allocate pinned host memory */
   hipHostMalloc(&found, 4);
   hipHostMalloc(&diff, 1);
   hipHostMalloc(&phash, 32);
   hipHostMalloc(&precompute_ctx, sizeof(SHA256_CTX));
   
   /* Copy immediate block data to pinned memory */
   *found = 0;
   *diff = difficulty;
   memcpy(phash, prevhash, 32);
   
   /* Precompute SHA256 */
   sha256_init(precompute_ctx);
   sha256_update(precompute_ctx, bt, 92);
   
   /* Initialize GPU data asynchronously */
   for (i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      
      /* Get the best block/thread configs */
      hipOccupancyMaxPotentialBlockSize(&ctx[i].nblock, &ctx[i].nthread,
                                         cuda_find_peach, 16, 1024);
      ctx[i].total_threads = ctx[i].nblock * ctx[i].nthread;
      
      /* Create Stream */
      hipStreamCreate(&ctx[i].stream);
      
      /* Allocate device memory */
      hipMalloc(&ctx[i].d_found, 4);
      hipMalloc(&ctx[i].d_nonce, 32);
      
      /* Allocate associated device-host memory */
      hipHostMalloc(&ctx[i].found, 4);
      hipHostMalloc(&ctx[i].nonce, 32);
      hipHostMalloc(&ctx[i].input, ctx[i].nthread * 16);
      
      /* Copy immediate block data to device memory */
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_difficulty), diff, 1, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_phash), phash, 32, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_precomputed_sha256), precompute_ctx,
                              sizeof(SHA256_CTX), 0, hipMemcpyHostToDevice,
                              ctx[i].stream);
      
      /* Set remaining device memory */
      hipMemsetAsync(ctx[i].d_found, 0, 4, ctx[i].stream);
      memset(ctx[i].found, 0, 4);
      
      /* Set scan offset to 1024*/
      ctx[i].scan_offset = ctx[i].nthread;
      
      /* Setup map and cache */
      hipMalloc(&ctx[i].d_map, MAP_LENGTH);
      cuda_build_map<<<4096, 256, 16, ctx[i].stream>>>(ctx[i].d_map);
   }
   
   /* Check for any GPU initialization errors */
   for(i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(ctx[i].stream);
      if(cudaCheckError("init_cuda_peach()", i, __FILE__))
         return -1;
   }

   return nGPU;
}

void free_cuda_peach() {
   int i;
   
   /* Free pinned host memory */
   hipHostFree(diff);
   hipHostFree(found);
   hipHostFree(phash);
   hipHostFree(precompute_ctx);
   
   /* Free GPU data */
   for (i = 0; i<nGPU; i++) {
      hipSetDevice(i);
      
      /* Destroy Stream */
      hipStreamDestroy(ctx[i].stream);
      
      /* Free device memory */
      hipFree(ctx[i].d_found);
      hipFree(ctx[i].d_nonce);
      hipFree(ctx[i].d_map);
      
      /* Free associated device-host memory */
      hipHostFree(ctx[i].found);
      hipHostFree(ctx[i].nonce);
      hipHostFree(ctx[i].input);
   }
}

extern byte *trigg_gen(byte *in);

__host__ void cuda_peach(byte *bt, uint32_t *hps, byte *runflag)
{
   int i, j, k;
   double tdiff;
   uint32_t shps;
   uint64_t lastnHaiku, nHaiku, ustart, uend;
   timeval nSeconds;
   time_t gpu_stats_time = time(NULL);
   
   gettimeofday(&nSeconds, NULL);
   for(nHaiku = 0; *runflag && *found == 0; ) {
      for (i=0; i<nGPU; i++) {
         /* Check if GPU has finished */
         hipSetDevice(i);
         if(hipStreamQuery(ctx[i].stream) == hipSuccess) {
            /* Obtain haiku/s calc data */
            gettimeofday(&(ctx[i].t_end), NULL);
            ustart = 1000000 * ctx[i].t_start.tv_sec + ctx[i].t_start.tv_usec;
            if (ustart > 0) {
               uend = 1000000 * ctx[i].t_end.tv_sec + ctx[i].t_end.tv_usec;
               tdiff = (uend - ustart) / 1000.0 / 1000.0;
            }
            gettimeofday(&(ctx[i].t_start), NULL);
            
            /* Check for a solved block */
            if(*ctx[i].found==1) { /* SOLVED A BLOCK! */
               hipMemcpy(ctx[i].nonce, ctx[i].d_nonce, 32, hipMemcpyDeviceToHost);
               memcpy(bt + 92, ctx[i].nonce, 32);
               *found = 1;
               break;
            }
            
            /* Init GPU data if necessary */
            if (ctx[i].scan_offset + ctx[i].nblock >= ctx[i].nthread) {
               /* Reset offset */
               ctx[i].scan_offset = 0;
               /* Generate random seed array data */
               for(j = 0, k = ctx[i].nthread * 16; j < k; j += 16)
                  trigg_gen(ctx[i].input + j);
               /* Send new seed array data */
               hipMemcpyToSymbolAsync(HIP_SYMBOL(c_input), ctx[i].input, k, 0,
                                       hipMemcpyHostToDevice, ctx[i].stream);
            }
            /* Start GPU round */
            cuda_find_peach<<<ctx[i].nblock,ctx[i].nthread,0,ctx[i].stream>>>
            (ctx[i].scan_offset, ctx[i].d_map, ctx[i].d_found, ctx[i].d_nonce);
            /* Retrieve GPU found status */
            hipMemcpyAsync(ctx[i].found, ctx[i].d_found, 4, hipMemcpyDeviceToHost);

            /* Add to haiku count */
            nHaiku += ctx[i].total_threads;
            ctx[i].scan_offset += ctx[i].nblock;
            
            /* Perform per GPU Haiku/s cacluation */
            if (ustart > 0) {
               ctx[i].hps_index = (ctx[i].hps_index + 1) % 3;
               ctx[i].hps[ctx[i].hps_index] = ctx[i].total_threads / tdiff;
               shps = 0;
               for (j = 0; j < 3; j++) {
                  shps += ctx[i].hps[j];
               }
               ctx[i].ahps = shps / 3;
            }
         }
         
         /* Waiting on GPU? ... */
         if(cudaCheckError("cuda_peach()", i, __FILE__)) {
            *runflag = 0;
            return;
         }
      }
      
      /* Print GPU stats and chill if waiting on all GPUs */
      if(lastnHaiku == nHaiku) {
         /* Print GPU stats every 5 seconds */
         if ( (time(NULL) - gpu_stats_time) > 5 ) {
            for (j = 0; j < nGPU; j++) {
               if (enable_nvml) {
                  uint32_t temp = 0;
                  uint32_t power = 0;
                  nvmlReturn_t r = nvmlDeviceGetTemperature(gpus[j].nvml_dev, NVML_TEMPERATURE_GPU, &temp);
                  if (r != NVML_SUCCESS) {
                     printf("nvmlDeviceGetTemperature failed: %s\n", nvmlErrorString(r));
                  }

                  r = nvmlDeviceGetPowerUsage(gpus[j].nvml_dev, &power);
                  if (r != NVML_SUCCESS) {
                     printf("nvmlDeviceGetPowerUsage Failed: %s\n", nvmlErrorString(r));
                  }
                  gpus[j].temp = temp;
                  gpus[j].power = power;

                  printf("GPU %d: %7d H/s, Temperature: %d C, Power: %6.2f W\n", j,
                        ctx[j].ahps, gpus[j].temp, gpus[j].power / 1000.0);
               } /* else {
                  printf("GPU %d: %7d H/s\n", j, ctx[j].ahps);
               } */
            }
            gpu_stats_time = time(NULL);
         }
         /* Chill for 1ms */
         usleep(1000);
      }
      else lastnHaiku = nHaiku;
   }
   
   /* Calculate Final Haiku/s */
   ustart = 1000000 * nSeconds.tv_sec + nSeconds.tv_usec;
   gettimeofday(&nSeconds, NULL);
   uend = 1000000 * nSeconds.tv_sec + nSeconds.tv_usec;
   tdiff = (uend - ustart) / 1000.0 / 1000.0;
   *hps = (uint32_t) (nHaiku / tdiff);
   
   /* Reset Miner Data */
   *found = 0;
   for (i=0; i<nGPU; i++) {
      hipSetDevice(i);
      memset(ctx[i].found, 0, 4);
      hipMemset(ctx[i].d_found, 0, 4);
      ctx[i].scan_offset = ctx[i].nthread;
   }
}


}
